#include "hip/hip_runtime.h"
#include "src/coo2csc.h"
#include "sys/time.h"
#include "src/FastLoad_CPU.h"
#include "src/FastLoad_GPU.h"
#include "src/CSCSpMV.h"
#include "src/csr2csc_cuda.h"
#include "src/formatTransform_GPU.h"
#include "src/ColSort_GPU.h"

#include "src/cusparse_cuda.h"
#include <iostream>

void init(double* v, long const N) {
    #pragma omp parallel for
    for (long i = 0; i < N; ++i) {
        v[i] = 1.f/(rand() % 1024);
    }
}

void sparsify(double* v, long const N, int sparsity) {
    #pragma omp parallel for
    for (long i = 0; i < N; ++i) {
        if ((rand() % 100) + 1 < sparsity) {
            v[i] = 0.f;
        }
    }
}

// method to generate a random csc sparse matrix
void generate_random_csc_matrix(int M, int N, double sparsity, double* csc_val, int* csc_rowidx, int* csc_ptr) {
    int nnz = 0;
    for (int i = 0; i < N; i++) {
        csc_ptr[i] = nnz;
        for (int j = 0; j < M; j++) {
            if ((rand() % 100) > sparsity) {
                csc_val[nnz] = static_cast<double>(rand()) / RAND_MAX;
                csc_rowidx[nnz] = j;
                nnz++;
            }
        }
    }
    csc_ptr[N] = nnz;
}

// CPU implementation of csc_spmv
void csc_spmv_cpu(int m, int n, int nnz, int *csc_ptr, int *csc_rowidx, double *csc_val, double *x, double *y) {
  memset(y, 0, m * sizeof(double));
  #pragma omp parallel for
  for (int i = 0; i < n; i++) {
      for (int j = csc_ptr[i]; j < csc_ptr[i + 1]; j++) {
          int rowidx = csc_rowidx[j];
          double val = csc_val[j];
          #pragma omp atomic
          y[rowidx] += val * x[i];
      }
  }
}


__global__ void CSC_SpMV_naive(int m, int n, int nnz, int *csc_ptr, int *csc_rowIdx, double *csc_val, double *x, double *y) 
{
    int global_id = blockIdx.x *blockDim.x + threadIdx.x;
    for (int i = global_id; i < n; i += blockDim.x * gridDim.x) 
    {
        double dotProduct = 0;
        const int col_start = csc_ptr[i];
        const int col_end = csc_ptr[i + 1];
        
        for (int j = col_start; j < col_end; j++) 
        {
            dotProduct = csc_val[j] * x[i];
            int rowidx_tmp = csc_rowIdx[j];
            atomicAdd(&y[rowidx_tmp], dotProduct);
        }  
    }
}

void csc_spmv(int m, int n, int nnz, int *csc_ptr, int *csc_rowidx, double *csc_val, double *x, double *y)
{
    int *d_csc_rowidx;
    int *d_csc_ptr;
    double *d_csc_val;
    double *d_x;
    double *d_y;
    int numSMs;
    int numTests = 100;

    hipMalloc((void **)&d_csc_rowidx, nnz * sizeof(int));
    hipMalloc((void **)&d_csc_ptr, (n+1) * sizeof(int));
    hipMalloc((void **)&d_csc_val, nnz * sizeof(double));
    hipMalloc((void **)&d_x, n * sizeof(double));
    hipMalloc((void **)&d_y, m * sizeof(double));

    timeval t1, t2;
    double time_memcpy_h2d = 0, time_memcpy_d2h = 0, time_kernel = 0;

    gettimeofday(&t1, NULL);
    hipMemcpy(d_csc_ptr, csc_ptr, (n + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csc_rowidx, csc_rowidx, (nnz) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csc_val, csc_val, (nnz) * sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(d_x, x, (n) * sizeof(double), hipMemcpyHostToDevice); 
    gettimeofday(&t2, NULL);
    
    time_memcpy_h2d = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;

    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    // dry run
    CSC_SpMV_naive<<<32 * numSMs, 64>>>(m, n, nnz, d_csc_ptr, d_csc_rowidx, d_csc_val, d_x, d_y);
    hipMemset(d_y, 0, m * sizeof(double));
    hipDeviceSynchronize();

    // make array to store the times
    double *times = (double *)malloc(numTests * sizeof(double));

    for (int i=0; i<numTests; i++) {      
      hipMemset(d_y, 0, m * sizeof(double));
      gettimeofday(&t1, NULL);
      CSC_SpMV_naive<<<32 * numSMs, 64>>>(m, n, nnz, d_csc_ptr, d_csc_rowidx, d_csc_val, d_x, d_y);
      hipDeviceSynchronize();
      gettimeofday(&t2, NULL);
      times[i] = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;
    }
    
    // find average kernel time
    for (int i=0; i<numTests; i++) {
      time_kernel += times[i];
    }
    time_kernel /= numTests;

    gettimeofday(&t1, NULL);
    hipMemcpy(y, d_y, m * sizeof(double), hipMemcpyDeviceToHost);
    gettimeofday(&t2, NULL);
    time_memcpy_d2h = (t2.tv_sec - t1.tv_sec) * 1000.0 + (t2.tv_usec - t1.tv_usec) / 1000.0;

    double gflops = 2 * (double)nnz * 1.0e-6 / time_kernel;

    printf("CSC SpMV Time: %f ms\n", time_memcpy_h2d + time_kernel + time_memcpy_d2h);
    printf("Memory H2D Time: %f ms\n", time_memcpy_h2d);
    printf("Kernel Time: %f ms\n", time_kernel);
    printf("Memory D2H Time: %f ms\n", time_memcpy_d2h);
    printf("GFLOPS: %f\n", gflops);

    hipFree(d_csc_ptr);
    hipFree(d_csc_rowidx);
    hipFree(d_csc_val);
    hipFree(d_x);
    hipFree(d_y);
}

int main(int argc, char ** argv)
{
  assert(argc == 4);

  // printout the matrix size
  long const M = atoi(argv[1]);
  long const N = atoi(argv[2]);
  printf("M: %ld, N: %ld\n", M, N);

  // get sparsity
  int const sparsity = atof(argv[3]);

  // initialize x and y
  double *x = (double *)malloc(N * sizeof(double));
  double *y = (double *)malloc(M * sizeof(double));
  memset(y, 0, M * sizeof(double));

  // initialize x
  init(x, N);

  int nnz = 0;

  // allocate memory for csc matrix
  double *csc_val = (double *)malloc(M * N * sizeof(double)); // over-allocate to handle sparsity
  int *csc_rowidx = (int *)malloc(M * N * sizeof(int)); // over-allocate to handle sparsity
  int *csc_ptr = (int *)malloc((N + 1) * sizeof(int));

  // generate random csc matrix
  generate_random_csc_matrix(M, N, sparsity, csc_val, csc_rowidx, csc_ptr);

  // get nnz
  nnz = csc_ptr[N];
  printf("nnz: %d\n\n", nnz);

  // call csc_spmv
  csc_spmv(M, N, nnz, csc_ptr, csc_rowidx, csc_val, x, y);

  // golden y
  double *y_golden = (double *)malloc(M * sizeof(double)); 
  memset(y_golden, 0, M * sizeof(double));
  
  // call csc_spmv_cpu
  csc_spmv_cpu(M, N, nnz, csc_ptr, csc_rowidx, csc_val, x, y_golden);

  // check result is correct
  int error_count_check = 0;
  float error_threshold = 0.0001;
  for (int i = 0; i < M; i++) {
      if (abs(y_golden[i] - y[i]) > error_threshold) {
          error_count_check++;
      }
  }

  if (error_count_check != 0) {
      printf("Error: %d\n", error_count_check);
  }

  //~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
  // FASTLOAD SECTION

  // allocate space for fastload y
  double *y_fastload = (double *)malloc(M * sizeof(double));
  memset(y_fastload, 0, M * sizeof(double));

  int *nnzpercol = (int *)malloc(N * sizeof(int));
  memset(nnzpercol, 0, sizeof(int) * N);

  // get nnzpercol
  for (int i = 0; i < N; i++) {
      nnzpercol[i] = csc_ptr[i + 1] - csc_ptr[i];
  }

  int *sortrowidx_tmp = (int *)malloc(sizeof(int)*nnz);
  memset(sortrowidx_tmp,0,sizeof(int)*nnz);
  double *sortval_tmp = (double *)malloc(sizeof(double)*nnz);
  memset(sortval_tmp,0,sizeof(double)*nnz);
  int *sortnnz_tmp= (int *)malloc(sizeof(int)*(N));
  memset(sortnnz_tmp,0,sizeof(int)*N);
  double *sortx = (double *)malloc(sizeof(double)*N);  
  memset(sortx,0,sizeof(double)*N);

  // sort columns
  double timeForSort = 0;
  // on GPU
  ColSort(timeForSort, M, N, nnz, nnzpercol, csc_ptr, csc_rowidx, csc_val, sortrowidx_tmp, sortval_tmp, sortnnz_tmp, x, sortx);

  printf("\nFastLoad Times:\n");

  // print time for sort
  printf("Time for sort: %f ms\n", timeForSort);

  int h_count;
  double timeFormatTran=0;
  double timeFortmatClas=0;

  slide_matrix *matrixA = (slide_matrix *)malloc(sizeof(slide_matrix));

  // on GPU
  formatTransform(timeFormatTran, timeFortmatClas, matrixA, sortrowidx_tmp, sortval_tmp, sortnnz_tmp, nnz, N, M, h_count);

  // print time for format transform
  printf("Time for format transform: %f ms\n", timeFormatTran);
  printf("Time for format classification: %f ms\n", timeFortmatClas);

  // print total pre-processing time
  printf("Total pre-processing time: %f ms\n", timeForSort + timeFormatTran + timeFortmatClas);

  free(nnzpercol);
  free(sortrowidx_tmp);
  free(sortval_tmp);
  free(sortnnz_tmp);

  // call fastload
  char *filename = "temp.mtx";
  FastLoad_spmv(filename, matrixA, nnz, M, N, sortx, y_fastload, y_golden);

  // free memory
  free(x);
  free(y);
  free(csc_val);
  free(csc_rowidx);
  free(csc_ptr);
  free(y_golden);
  free(y_fastload);

  return 0;
}